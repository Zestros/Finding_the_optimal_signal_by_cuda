#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>
#include <cmath>
#include <fstream>
#include <string>


using namespace std;

__global__ void optimized_akf_kernel_with_generation(int offset, int* dev_max, size_t n)
{
    const size_t idx = blockIdx.x;           // Индекс блока (номер сигнала)
    const size_t tid = threadIdx.x;          // Номер потока внутри блока

    extern __shared__ int shared_mem[];      // Разделяемая память для промежуточных результатов

    // Генерируем сигнал непосредственно на GPU
    size_t unique_signal_idx = idx + offset;
    size_t signal = unique_signal_idx;       // Уникальный сигнал

    // Расчёт корреляционного значения для каждой позиции АКФ
    int akf_value = 0;
    for (size_t j = 0; j < n; j++) {
        if (tid + j < n) {                   // Проверка границы
            bool bit_i = (signal >> (tid + j)) & 1;
            bool bit_j = (signal >> j) & 1;
            akf_value += (bit_i ^ bit_j) ? -1 : 1; // XOR для проверки равенства битов
        }
    }

    // Сохраняем результат в разделяемую память
    shared_mem[tid] = akf_value;
    __syncthreads();

    // Поиск максимума второго порядка среди всех позиций АКФ
    int max1_val = INT_MIN;
    int max2_val = INT_MIN;

    for (size_t i = 0; i < n; ++i) {
        if (abs(shared_mem[i]) > max1_val) {
            max2_val = max1_val;
            max1_val = abs(shared_mem[i]);
        }
        else if (abs(shared_mem[i]) > max2_val && abs(shared_mem[i]) != max1_val) {
            max2_val = abs(shared_mem[i]);
        }
    }

    // Результат записывается обратно в глобальную память
    dev_max[idx] = max2_val;
}

int main()
{
    size_t n = 29;
    size_t N = 1048576;                     // Пакет сигналов
    size_t NM = (1 << n) / N;                // Число пакетов
    int maxD = 10000000;
    size_t bestSignal = 0;

    int* dev_max;
    hipMalloc((void**)&dev_max, N * sizeof(int)); // Память для результирующих данных

    // Профилировка времени
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (size_t k = 0; k < NM; k++)
    {
        // Запуск ядра с прямым созданием сигналов на GPU
        dim3 threadsPerBlock(n);              // Потоки = длина сигнала
        dim3 blocksPerGrid(N);                // Блока на каждый сигнал пакета

        optimized_akf_kernel_with_generation << <blocksPerGrid, threadsPerBlock, n * sizeof(int) >> > (k * N, dev_max, n);

        // Получаем результаты обратно на хост
        int* maxs = new int[N];               // Временный буфер
        hipMemcpy(maxs, dev_max, N * sizeof(int), hipMemcpyDeviceToHost);

        for (size_t i = 0; i < N; i++)
        {
            if (maxs[i] < maxD)
            {
                maxD = maxs[i];
                bestSignal = i + k * N;
            }
        }
        delete[] maxs;
    }

    // Завершаем замер времени
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsed_time_ms;
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    printf("GPU execution time: %.3f seconds\n", elapsed_time_ms / 1000);

    // Финальный вывод
    cout << "Best signal: " << invertBinaryString(intToBinaryString(bestSignal, n)) << endl;

    hipFree(dev_max);
    return 0;
}

